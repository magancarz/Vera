#include "Shape.h"

__device__ Shape::Shape(Object* parent, size_t id, Material* material)
    : parent(parent), id(id), material(material) {}

float Shape::calculatePDFValueOfEmittedLight(const glm::vec3& origin, const glm::vec3& direction)
{
    return 0;
}

__device__ void Shape::setTransform(glm::mat4* object_to_world_val, glm::mat4* world_to_object_val)
{
    this->object_to_world = object_to_world_val;
    this->world_to_object = world_to_object_val;
    calculateObjectBounds();
    calculateWorldBounds();
    applyTransform(*this->object_to_world);
    calculateShapeSurfaceArea();
}

__device__ void Shape::resetTransform()
{
    if (world_to_object != nullptr)
    {
        applyTransform(*world_to_object);
    }
}

__device__ glm::vec3 Shape::randomDirectionAtShape(hiprandState* curand_state, const glm::vec3& origin)
{
    return {0.f, 0.f, 0.f};
}

__device__ bool Shape::isEmittingLight() const
{
    return false;
}

float Shape::scatteringPDF(const HitRecord* rec, const Ray* scattered) const
{
    return 0.f;
}

glm::vec3 Shape::emitted(const glm::vec2& uv)
{
    return {0, 0, 0};
}
