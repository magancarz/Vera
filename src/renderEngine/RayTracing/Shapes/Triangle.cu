#include "hip/hip_runtime.h"
#include "Triangle.h"

#include "Models/TriangleData.h"
#include "Materials/Material.h"
#include "Utils/CurandUtils.h"

__device__ Triangle::Triangle(Object* parent, size_t id, Material* material, const TriangleData& triangle_data)
    : Shape(parent, id, material)
{
    object_to_world = nullptr;
    world_to_object = nullptr;

    x = triangle_data.vertices[0];
    y = triangle_data.vertices[1];
    z = triangle_data.vertices[2];

    computeAverageNormal();
}

__device__ HitRecord Triangle::checkRayIntersection(const Ray* r) const
{
    constexpr float EPSILON = 0.00000001f;
    constexpr float MIN_DST = 0.0001f;
    glm::vec3 edge_xy = y.position - x.position;
    glm::vec3 edge_xz = z.position - x.position;
    glm::vec3 ao = r->origin - x.position;
    glm::vec3 normal = cross(edge_xy, edge_xz);

    float determinant = dot(-r->direction, normal);
    if (fabs(determinant) < EPSILON) return {};
    float inv_det = 1.f / determinant;

    float dst = dot(ao, normal) * inv_det;
    float u = dot(-r->direction, cross(ao, edge_xz)) * inv_det;
    float v = dot(-r->direction, cross(edge_xy, ao)) * inv_det;
    float w = 1.f - u - v;

    HitRecord hit_record_result{};
    if (dst >= MIN_DST && u >= 0 && v >= 0 && w >= 0)
    {
        hit_record_result.uv = w * x.texture_coordinate + u * y.texture_coordinate + v * z.texture_coordinate;
        if (material->getColorAlphaValue(hit_record_result.uv) < 0.5f)
        {
            return hit_record_result;
        }
        hit_record_result.hit_point = r->origin + r->direction * dst;
        hit_record_result.front_face = determinant > 0;
        hit_record_result.t = dst;
        hit_record_result.did_hit_anything = true;
        hit_record_result.parent_object = parent;
        hit_record_result.triangle_id = id;
        hit_record_result.material = material;
        hit_record_result.color = material->getColor(hit_record_result.uv);
        glm::vec3 new_normal;
    	if (material->hasNormalMap())
        {
            new_normal = w * material->getNormal(x.texture_coordinate) + u * material->getNormal(y.texture_coordinate) + v * material->getNormal(z.texture_coordinate);
        }
    	else
        {
            new_normal = getNormalAt(w, u, v);
        }
        hit_record_result.normal = determinant > 0 ? normalize(new_normal) : normalize(-new_normal);
    }

    return hit_record_result;
}

__device__ float Triangle::calculatePDFValueOfEmittedLight(const glm::vec3& origin, const glm::vec3& direction)
{
    Ray ray{origin, direction};
    const auto rec = checkRayIntersection(&ray);
    if (rec.triangle_id != id)
    {
        return 0;
    }

    float cosine = fabs(dot(direction, rec.normal));
    const float distance_squared = rec.t * rec.t;
    cosine = cosine < 0.00000001f ? 0.00000001f : cosine;

    return distance_squared / (cosine * area);
}

__device__ glm::vec3 Triangle::randomDirectionAtShape(hiprandState* curand_state, const glm::vec3& origin)
{
    float u = randomFloat(curand_state);
    float v = randomFloat(curand_state);

    if (u + v > 1.f)
    {
        u = 1.f - u;
        v = 1.f - v;
    }

    const float w = 1.f - u - v;

    const glm::vec3 random_point_on_a_triangle = u * x.position + v * y.position + w * z.position;

    return glm::normalize(random_point_on_a_triangle - origin);
}

__device__ void Triangle::applyTransform(const glm::mat4& transform)
{
    x.position = glm::vec3(transform * glm::vec4(x.position, 1.0f));
    y.position = glm::vec3(transform * glm::vec4(y.position, 1.0f));
    z.position = glm::vec3(transform * glm::vec4(z.position, 1.0f));
    transformNormal(transform);
}

__device__ void Triangle::calculateObjectBounds()
{
    object_bounds = boundsFromUnion(x.position, y.position, z.position);
}

__device__ void Triangle::calculateWorldBounds()
{
    const glm::vec3 world_x = glm::vec3(*object_to_world * glm::vec4(x.position, 1.0f));
    const glm::vec3 world_y = glm::vec3(*object_to_world * glm::vec4(y.position, 1.0f));
    const glm::vec3 world_z = glm::vec3(*object_to_world * glm::vec4(z.position, 1.0f));
    world_bounds = boundsFromUnion(world_x, world_y, world_z);
}

__device__ void Triangle::calculateShapeSurfaceArea()
{
    area = glm::length(glm::cross(glm::vec3{y.position - x.position}, glm::vec3{z.position - x.position})) / 2.f;
}

__host__ __device__ bool Triangle::isEmittingLight() const
{
    return material->getSpecularValue(x.texture_coordinate).g > 0.5f ||
        material->getSpecularValue(y.texture_coordinate).g > 0.5f ||
        material->getSpecularValue(z.texture_coordinate).g > 0.5f;
}

__host__ __device__ glm::vec3 Triangle::getNormalAt(const glm::vec3& barycentric_coordinates) const
{
    return x.normal * barycentric_coordinates.x + y.normal * barycentric_coordinates.y + z.normal * barycentric_coordinates.z;
}

__host__ __device__ glm::vec3 Triangle::getNormalAt(float u, float v, float w) const
{
    return x.normal * u + y.normal * v + z.normal * w;
}

__device__ void Triangle::computeAverageNormal()
{
    if (!areTriangleNormalsValid())
    {
        average_normal = normalize(glm::cross(y.position - x.position, z.position - x.position));
        x.normal = y.normal = z.normal = average_normal;
        return;
    }

    constexpr float one_third = 1.f / 3;
    average_normal = getNormalAt(one_third, one_third, one_third);
}

__device__ bool Triangle::areTriangleNormalsValid() const
{
    return glm::length(x.normal) < 0.0001f && glm::length(y.normal) < 0.0001f && glm::length(z.normal) < 0.0001f;
}

__device__ void Triangle::transformNormal(const glm::mat4& transform)
{
    auto transposed_inverse = glm::mat3{transform};
    transposed_inverse = glm::transpose(glm::inverse(transposed_inverse));

    x.normal = glm::normalize(transposed_inverse * x.normal);
    y.normal = glm::normalize(transposed_inverse * y.normal);
    z.normal = glm::normalize(transposed_inverse * z.normal);
    average_normal = glm::normalize(transposed_inverse * average_normal);
}
