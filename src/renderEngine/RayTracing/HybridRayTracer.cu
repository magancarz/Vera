#include "hip/hip_runtime.h"
#include "HybridRayTracer.h"

#include <>
#include <cuda/std/chrono>

#include <cuda/hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <chrono>

#include "RayTracerCamera.h"
#include "Scene/Scene.h"
#include "Objects/Camera.h"
#include "Utils/DeviceMemoryPointer.h"

#include "ScatterRecord.h"
#include "PDF/HittablePDF.h"
#include "PDF/MixturePDF.h"
#include "Materials/Material.h"
#include "Utils/CurandUtils.h"
#include "IntersectionAccelerators/BVHTreeTraverser.h"

namespace RayTracing
{
    __device__ glm::vec3 gatherColorInformationFromSceneIntersectionWithHybridRayTracing(BVHTreeTraverser* intersection_accelerator_tree_traverser, Triangle** sampled_shapes, int num_of_sampled_shapes, Ray* ray, int depth)
    {
        HittablePDF hittable_pdf(ray->curand_state, intersection_accelerator_tree_traverser, sampled_shapes, num_of_sampled_shapes);
        glm::vec3 color{1.f};
        int max_iterations = 50;
        max_iterations = depth > max_iterations ? depth * 2 : max_iterations;
        for (int current_depth = depth, current_iteration = 0; current_depth >= 0 && current_iteration < max_iterations; --current_depth, ++current_iteration)
        {
            if (depth == 0)
            {
                return glm::vec3{0};
            }

            HitRecord rec = intersection_accelerator_tree_traverser->checkIntersection(ray);
            if (rec.did_hit_anything)
            {
                ScatterRecord scatter_record{};
                if (rec.material->scatter(ray, &rec, &scatter_record))
                {
                    if (scatter_record.is_specular)
                    {
                        scatter_record.specular_ray.curand_state = ray->curand_state;
                        *ray = scatter_record.specular_ray;
                        color *= rec.color;
                        ++current_depth;
                        continue;
                    }

                    hittable_pdf.changeHitRecord(&rec);
                    MixturePDF mixture_pdf{
                        ray->curand_state,
                        hittable_pdf,
                        scatter_record.pdf};
                    Ray scattered{rec.hit_point, mixture_pdf.generate()};
                    auto pdf = mixture_pdf.value(scattered.direction);
                    auto scattering_pdf = rec.material->scatteringPDF(&rec, &scattered);

                    scattered.curand_state = ray->curand_state;
                    *ray = scattered;
                    color *= rec.color * scattering_pdf / pdf;
                    continue;
                }

                return color * rec.material->emitted(rec.uv);
            }

            float t = 0.5f * (ray->direction.y + 1.f);
            return (1.f - t) * glm::vec3{1} + t * glm::vec3{0.5f, 0.7f, 1.0f};
        }

        return glm::vec3{0.f};
    }

    __global__ void generateImageWithHybridRayTracing(unsigned char* cuda_texture_array, unsigned long* accumulated_texture_data, int number_of_total_samples, RayTracerCamera* camera, BVHTreeTraverser* intersection_accelerator_tree_traverser, Triangle** sampled_shapes, int num_of_sampled_shapes, int depth, int image_width, int image_height)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < image_width && y < image_height)
        {
            constexpr int samples_per_pixel = 5;
            const auto image_width_reciprocal = 1.f / static_cast<float>(image_width);
            const auto image_height_reciprocal = 1.f / static_cast<float>(image_height);

            auto curand_state = new hiprandState();
            hiprand_init(cuda::std::chrono::high_resolution_clock::now().time_since_epoch().count(), x + y * blockDim.x, 0, curand_state);

            glm::vec3 color{0};
            for (int i = 0; i < samples_per_pixel; ++i)
            {
                const float u = (static_cast<float>(x) + hiprand_uniform(curand_state) - 0.5f) * image_width_reciprocal;
                const float v = (static_cast<float>(image_height - y) + hiprand_uniform(curand_state) - 0.5f) * image_height_reciprocal;
                Ray ray = camera->getRay(curand_state, u, v);
                ray.curand_state = curand_state;
                color += gatherColorInformationFromSceneIntersectionWithHybridRayTracing(intersection_accelerator_tree_traverser, sampled_shapes, num_of_sampled_shapes, &ray, depth);
            }

            delete curand_state;

            color /= static_cast<float>(samples_per_pixel);
            color = glm::clamp(color, 0.f, 1.f);

            const size_t index = y * image_width + x;

            const auto ir = static_cast<unsigned char>(255.99f * color.x);
            const auto ig = static_cast<unsigned char>(255.99f * color.y);
            const auto ib = static_cast<unsigned char>(255.99f * color.z);

            accumulated_texture_data[index * 3 + 0] += ir;
            accumulated_texture_data[index * 3 + 1] += ig;
            accumulated_texture_data[index * 3 + 2] += ib;
            
            cuda_texture_array[index * 3 + 0] = static_cast<float>(accumulated_texture_data[index * 3 + 0]) / static_cast<float>(number_of_total_samples);
            cuda_texture_array[index * 3 + 1] = static_cast<float>(accumulated_texture_data[index * 3 + 1]) / static_cast<float>(number_of_total_samples);
            cuda_texture_array[index * 3 + 2] = static_cast<float>(accumulated_texture_data[index * 3 + 2]) / static_cast<float>(number_of_total_samples);
        }
    }
}

void HybridRayTracer::runRayTracer(Scene* scene, const std::shared_ptr<RayTracedImage>& current_image, const dim3& blocks, const dim3& threads_per_block)
{
    RayTracing::generateImageWithHybridRayTracing<<<blocks, threads_per_block>>>(
        current_image->texture_data_ptr,
        current_image->accumulated_texture_data.data(),
        current_image->generated_samples,
        cuda_camera.data(),
        scene->intersection_accelerator_tree_traverser.data(),
        scene->scene_light_sources.data(),
        scene->scene_light_sources.size(),
        current_image->image_config.number_of_ray_bounces,
        current_image->image_config.image_width,
        current_image->image_config.image_height);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}
