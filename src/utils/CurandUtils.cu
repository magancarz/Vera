#include "hip/hip_runtime.h"
#include "CurandUtils.h"

#include <hiprand/hiprand_kernel.h>

__global__ void initCurandState(hiprandState* curand_state, unsigned long long seed)
{
    hiprand_init(seed, 0, 0, curand_state);
}